#include "hip/hip_runtime.h"
#include <iostream>
#include <utility>
#include <cmath>
#include <cuda/hip/hip_runtime_api.h>

extern "C" {
#include <matrix/format.h>
}

void ensure_device_capabilities_csr(const std::pair<dim3, dim3>& dims, const hipDeviceProp_t& device_props) {
    if(dims.first.x > device_props.maxGridSize[0]) {
        std::cerr 
            << "device is unable to handle " 
            << dims.first.x 
            << " grid dimensionality (x-axis)."
            << " Max allowed is "
            << device_props.maxGridSize[0]
            << std::endl;
        exit(EXIT_FAILURE);
    }

    if(dims.second.x > device_props.maxThreadsPerBlock) {
        std::cerr 
            << "device is unable to handle " 
            << dims.second.x 
            << " block dimensionality (x-axis)."
            << " Max allowed is "
            << device_props.maxThreadsPerBlock
            << std::endl;
        exit(EXIT_FAILURE);
    }
}

void ensure_device_capabilities_csr(const std::tuple<dim3, dim3, size_t>& dims, const hipDeviceProp_t& device_props) {
    ensure_device_capabilities_csr(std::make_pair<>(std::get<0>(dims), std::get<1>(dims)), device_props);

    size_t shmem_size = std::get<2>(dims);
    if(shmem_size > device_props.sharedMemPerBlock) {
        std::cerr
            << "device is unable to handle "
            << shmem_size
            << "B of shared memory amount per block."
            << " Max allowed is "
            << device_props.sharedMemPerBlock
            << std::endl;
        exit(EXIT_FAILURE);
    }
}

std::pair<dim3, dim3> get_dims_for_csr_v1(int nrows, const hipDeviceProp_t& device_props) {
    int max_thr_per_blk = device_props.maxThreadsPerBlock;

    if(nrows <= max_thr_per_blk) {
        dim3 grid_dim(1, 1);
        dim3 block_dim(nrows, 1);
        return std::make_pair<>(grid_dim, block_dim);
    }

    double splsz = nrows / max_thr_per_blk;
    int xgridsz = nrows % max_thr_per_blk ? ceil(splsz) + 1 : splsz;

    dim3 grid_dim(xgridsz, 1);
    dim3 block_dim(max_thr_per_blk, 1);
    return std::make_pair<>(grid_dim, block_dim);
}

__global__ void __kernel_csr_v1(
        const uint64_t *irp, 
        const uint64_t *ja, 
        const double *as, 
        uint32_t m, 
        const double *x, 
        double *y) {

    const int thread_global_index = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread_global_index >= m) {
        return;
    }

    double t = 0;
    for(int j = irp[thread_global_index]; j < irp[thread_global_index + 1]; j++) {
        t += as[j] * x[ja[j]];
    }

    y[thread_global_index] = t;
}

std::pair<dim3, dim3> get_dims_for_csr_v2(int nrows, const hipDeviceProp_t& device_props) {
    int warp_size = device_props.warpSize;
    int max_thr_per_blk = device_props.maxThreadsPerBlock;

    if(nrows * warp_size <= max_thr_per_blk) {
        dim3 grid_dim(1,1);
        dim3 block_dim(nrows * warp_size, 1);
        return std::make_pair<>(grid_dim, block_dim);
    }

    double splsz = nrows * warp_size / max_thr_per_blk;
    int xgridsz = (nrows * warp_size) % max_thr_per_blk ? ceil(splsz) + 1 : splsz;

    dim3 grid_dim(xgridsz, 1);
    dim3 block_dim(max_thr_per_blk, 1);
    return std::make_pair<>(grid_dim, block_dim);
}

__global__ void __kernel_csr_v2(
        const uint64_t *irp, 
        const uint64_t *ja, 
        const double *as, 
        uint32_t m, 
        const double *x, 
        double *y) {

    if(threadIdx.x % warpSize != 0) {
        return;
    }

    const int warp_global_index = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    
    if(warp_global_index >= m) {
        return;
    }

    double t = 0;

    for(int j = irp[warp_global_index]; j < irp[warp_global_index + 1]; j++) {
        t += as[j] * x[ja[j]];
    }

    y[warp_global_index] = t;
}

std::tuple<dim3, dim3, size_t> get_dims_for_csr_v3(int nrows, const hipDeviceProp_t& device_props) {
    auto csrv2dims = get_dims_for_csr_v2(nrows, device_props);
    size_t shmemsize = csrv2dims.second.x * sizeof(double); 
    return std::make_tuple<>(csrv2dims.first, csrv2dims.second, shmemsize);
}

__global__ void __kernel_csr_v3(
        const uint64_t *irp, 
        const uint64_t *ja, 
        const double *as, 
        int32_t m, 
        const double *x, 
        double *y) {

    extern __shared__ double row_shmem[];

    const int warp_global_index = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if(warp_global_index >= m) {
        return;
    }

    const int irps = irp[warp_global_index];
    const int irpe = irp[warp_global_index + 1];
    const int nj = irpe - irps;

    const int thread_idx_in_warp = threadIdx.x % warpSize;

    row_shmem[threadIdx.x] = 0;

    if(nj <= warpSize) {
        if(thread_idx_in_warp > nj + 1) {
            return;
        }

        const int j = warp_global_index + thread_idx_in_warp;

        if(irps <= j && j < irpe) {
            row_shmem[threadIdx.x] = as[j] * x[ja[j]];
        }
    } else {
        for(int i = thread_idx_in_warp; i < nj; i += warpSize) {
            const int j = warp_global_index + i;
            if(irps <= j && j < irpe) {
                row_shmem[threadIdx.x] += as[j] * x[ja[j]];
            }
        }
    }

    __syncthreads();

    if(thread_idx_in_warp == 0) {
        const int warp_local_index = threadIdx.x / warpSize;
        for(int i = 0; i < warpSize; i++) {
            y[warp_global_index] += row_shmem[warp_local_index * warpSize + i];
        }
    }
}

int main() {

    int device_id;
    checkCudaErrors(hipGetDevice(&device_id));

    hipDeviceProp_t device_props;
    checkCudaErrors(hipGetDeviceProperties(&device_props, device_id));

    hipDeviceReset();

    int m = 48;
    int n = 48;

    struct coo_format coo[] = {
        { .i = 0, .j = 0, .v = 1 },
        { .i = 1, .j = 1, .v = 2 },
        { .i = 2, .j = 2, .v = 3 },
        { .i = 3, .j = 3, .v = 4 },
        { .i = 4, .j = 4, .v = 5 },
        { .i = 5, .j = 5, .v = 6 },
        { .i = 6, .j = 6, .v = 1 },
        { .i = 7, .j = 7, .v = 2 },
        { .i = 8, .j = 8, .v = 3 },
        { .i = 9, .j = 9, .v = 4 },
        { .i = 10, .j = 10, .v = 5 },
        { .i = 11, .j = 11, .v = 6 },
        { .i = 12, .j = 12, .v = 1 },
        { .i = 13, .j = 13, .v = 2 },
        { .i = 14, .j = 14, .v = 3 },
        { .i = 15, .j = 15, .v = 4 },
        { .i = 16, .j = 16, .v = 5 },
        { .i = 17, .j = 17, .v = 6 },
        { .i = 18, .j = 18, .v = 1 },
        { .i = 19, .j = 19, .v = 2 },
        { .i = 20, .j = 20, .v = 3 },
        { .i = 21, .j = 21, .v = 4 },
        { .i = 22, .j = 22, .v = 5 },
        { .i = 23, .j = 23, .v = 6 },
        { .i = 24, .j = 24, .v = 1 },
        { .i = 25, .j = 25, .v = 2 },
        { .i = 26, .j = 26, .v = 3 },
        { .i = 27, .j = 27, .v = 4 },
        { .i = 28, .j = 28, .v = 5 },
        { .i = 29, .j = 29, .v = 6 },
        { .i = 30, .j = 30, .v = 1 },
        { .i = 31, .j = 31, .v = 2 },
        { .i = 32, .j = 32, .v = 3 },
        { .i = 33, .j = 33, .v = 4 },
        { .i = 34, .j = 34, .v = 5 },
        { .i = 35, .j = 35, .v = 6 },
        { .i = 36, .j = 36, .v = 1 },
        { .i = 37, .j = 37, .v = 2 },
        { .i = 38, .j = 38, .v = 3 },
        { .i = 39, .j = 39, .v = 4 },
        { .i = 40, .j = 40, .v = 5 },
        { .i = 41, .j = 41, .v = 6 },
        { .i = 42, .j = 42, .v = 1 },
        { .i = 43, .j = 43, .v = 2 },
        { .i = 44, .j = 44, .v = 3 },
        { .i = 45, .j = 45, .v = 4 },
        { .i = 46, .j = 46, .v = 5 },

        { .i = 47, .j = 47, .v = 1 },
        { .i = 47, .j = 46, .v = 1 },
        { .i = 47, .j = 45, .v = 1 },
        { .i = 47, .j = 44, .v = 1 },
        { .i = 47, .j = 43, .v = 1 },
        { .i = 47, .j = 42, .v = 1 },
        { .i = 47, .j = 41, .v = 1 },
        { .i = 47, .j = 40, .v = 1 },
        { .i = 47, .j = 39, .v = 1 },
        { .i = 47, .j = 38, .v = 1 },
        { .i = 47, .j = 37, .v = 1 },
        { .i = 47, .j = 36, .v = 1 },
        { .i = 47, .j = 35, .v = 1 },
        { .i = 47, .j = 34, .v = 1 },
        { .i = 47, .j = 33, .v = 1 },
        { .i = 47, .j = 32, .v = 1 },
        { .i = 47, .j = 31, .v = 1 },
        { .i = 47, .j = 30, .v = 1 },
        { .i = 47, .j = 29, .v = 1 },
        { .i = 47, .j = 28, .v = 1 },
        { .i = 47, .j = 27, .v = 1 },
        { .i = 47, .j = 26, .v = 1 },
        { .i = 47, .j = 25, .v = 1 },
        { .i = 47, .j = 24, .v = 1 },
        { .i = 47, .j = 23, .v = 1 },
        { .i = 47, .j = 22, .v = 1 },
        { .i = 47, .j = 21, .v = 1 },
        { .i = 47, .j = 20, .v = 1 },
        { .i = 47, .j = 19, .v = 1 },
        { .i = 47, .j = 18, .v = 1 },
        { .i = 47, .j = 17, .v = 1 },
        { .i = 47, .j = 16, .v = 1 },
        { .i = 47, .j = 15, .v = 1 },
        { .i = 47, .j = 14, .v = 1 },
    };

    double host_x[48];

    memset(host_x, 0, sizeof(host_x));

    host_x[47] = 1;
    host_x[46] = 1;
    host_x[45] = 1;
    host_x[44] = 1;
    host_x[43] = 1;
    host_x[42] = 1;
    host_x[41] = 1;
    host_x[40] = 1;
    host_x[39] = 1;
    host_x[38] = 1;
    host_x[37] = 1;
    host_x[36] = 1;
    host_x[35] = 1;
    host_x[34] = 1;
    host_x[33] = 1;
    host_x[32] = 1;
    host_x[31] = 1;
    host_x[30] = 1;
    host_x[29] = 1;
    host_x[28] = 1;
    host_x[27] = 1;
    host_x[26] = 1;
    host_x[25] = 1;
    host_x[24] = 1;
    host_x[23] = 1;
    host_x[22] = 1;
    host_x[21] = 1;
    host_x[20] = 1;
    host_x[19] = 1;
    host_x[18] = 1;
    host_x[17] = 1;
    host_x[16] = 1;
    host_x[15] = 1;
    host_x[14] = 1;

    /* ---- */

    int nz = sizeof(coo) / sizeof(struct coo_format);
    struct csr_format csr;
    coo_to_csr(&csr, coo, nz, m);

    uint64_t *dev_irp;
    uint64_t *dev_ja;
    double *dev_as;
    double *dev_y;
    double *dev_x;

    checkCudaErrors(hipMalloc(&dev_irp, sizeof(uint64_t) * (m + 1)));
    checkCudaErrors(hipMalloc(&dev_ja, sizeof(uint64_t) * (nz)));
    checkCudaErrors(hipMalloc(&dev_as, sizeof(double) * (nz)));
    checkCudaErrors(hipMalloc(&dev_y, sizeof(double) * (m)));
    checkCudaErrors(hipMalloc(&dev_x, sizeof(host_x)));

    checkCudaErrors(hipMemcpy(dev_irp, csr.irp, sizeof(uint64_t) * (m + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_ja, csr.ja, sizeof(uint64_t) * (nz), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_as, csr.as, sizeof(double) * (nz), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_x, host_x, sizeof(host_x), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(dev_y, 0, sizeof(double) * (m)));

    auto csr_v3_dims = get_dims_for_csr_v3(m, device_props);
    ensure_device_capabilities_csr(csr_v3_dims, device_props);

    hipEvent_t start;
    hipEvent_t stop;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, 0));
    __kernel_csr_v3<<<std::get<0>(csr_v3_dims), std::get<1>(csr_v3_dims), std::get<2>(csr_v3_dims)>>>(dev_irp, dev_ja, dev_as, m, dev_x, dev_y);
    checkCudaErrors(hipEventRecord(stop, 0));

    checkCudaErrors(hipDeviceSynchronize());

    float timeMs;
    checkCudaErrors(hipEventElapsedTime(&timeMs, start, stop));
    std::cout << timeMs / 1000 << " s" << std::endl;

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    /*
    auto csr_v1_dims = get_dims_for_csr_v1(m, device_props);
    ensure_device_capabilities_csr(csr_v1_dims, device_props);
    __kernel_csr_v1<<<csr_v1_dims.first,csr_v1_dims.second>>>(dev_irp, dev_ja, dev_as, m, dev_x, dev_y);
    checkCudaErrors(hipDeviceSynchronize());

    auto csr_v2_dims = get_dims_for_csr_v2(m, device_props);
    ensure_device_capabilities_csr(csr_v2_dims, device_props);
    __kernel_csr_v2<<<csr_v2_dims.first,csr_v2_dims.second>>>(dev_irp, dev_ja, dev_as, m, dev_x, dev_y);
    checkCudaErrors(hipDeviceSynchronize());
    */


    checkCudaErrors(hipFree(dev_irp));
    checkCudaErrors(hipFree(dev_ja));
    checkCudaErrors(hipFree(dev_as));
    checkCudaErrors(hipFree(dev_x));

    double host_y[m];

    checkCudaErrors(hipMemcpy(host_y, dev_y, sizeof(double) * (m), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(dev_y));

    puts("");
    for(int i = 0; i < m; i++) {
        printf("y[%d] = %lg\n", i, host_y[i]);
    }
    puts("");

    return 0;
}
